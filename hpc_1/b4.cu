
#include <hip/hip_runtime.h>
#include<iostream>
#include<cstdio>

using namespace std;


__global__ void var(int *a,int *b,int n,float mean)
{


	int block=256*blockIdx.x;
	float sum=0;


	for(int i=block;i<min(block+256,n);i++)
	{


	sum=sum+(a[i]-mean)*(a[i]-mean);


	}
	b[blockIdx.x]=sum;

}




__global__ void sum(int *a,int *b,int n)
{


	int block=256*blockIdx.x;
	int sum=0;


	for(int i=block;i<min(block+256,n);i++)
	{


	sum=sum+a[i];


	}
	b[blockIdx.x]=sum;

}


int main()
{


cout<<"Enter the no of elements"<<endl;
int n;
cin>>n;


int a[n];

cout<<"Enter the elements for the array"<<endl;

	for(int i=0;i<n;i++)
	{

		cin>>a[i];
	}


int *ad,*bd;

int size=n*sizeof(int);


hipMalloc(&ad,size);
hipMemcpy(ad,a,size,hipMemcpyHostToDevice);


int grids=ceil(n*1.0f/256.0f);

hipMalloc(&bd,grids*sizeof(int));

dim3 grid(grids,1);
dim3 block(1,1);


int p=n;


hipEvent_t start,end;

hipEventCreate(&start);
hipEventCreate(&end);

hipEventRecord(start);



while(n>1)
{

	sum<<<grid,block>>>(ad,bd,n);
	n=ceil(n*1.0f/256.0f);
	hipMemcpy(ad,bd,n*sizeof(int),hipMemcpyDeviceToDevice);

}


hipEventRecord(end);
hipEventSynchronize(end);


float time=0;


hipEventElapsedTime(&time,start,end);


cout<<"The time is"<<time<<endl;




int add[2];


n=p;


hipMemcpy(add,ad,4,hipMemcpyDeviceToHost);


cout<<"The sum is  "<<add[0]<<endl;

float mean=0.0f;

mean=add[0]/(n*1.0f);

cout<<"The mean is   "<<mean<<endl;



////////////////////sd part


hipMalloc(&ad,size);
hipMemcpy(ad,a,size,hipMemcpyHostToDevice);

hipMalloc(&bd,grids*sizeof(int));

var<<<grid,block>>>(ad,bd,n,mean);
n=ceil(n*1.0f/256.0f);	




sum<<<grid,block>>>(bd,ad,n);



hipMemcpy(add,ad,4,hipMemcpyDeviceToHost);

float sd=sqrt(add[0]/p*1.0f);

cout<<"The standard deviation is "<<sd<<endl;


}
